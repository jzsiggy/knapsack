#include "hip/hip_runtime.h"
// Include the necessary libraries
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

// Define a Movie struct which contains start, end, and category of a movie
struct Movie {
  int start;
  int end;
  int category;
};

// Define a struct that applies the schedule calculation logic on each combination
struct ScheduleFunctor {
  int movieCount;  // Number of movies
  int categoryCount;  // Number of categories
  int *categoryLimits;  // Limits for each category
  Movie *movies;  // List of movies

  // ScheduleFunctor constructor
  ScheduleFunctor(int movieCount, Movie *movies, int categoryCount, int *categoryLimits)
      : movieCount(movieCount), movies(movies), categoryCount(categoryCount), categoryLimits(categoryLimits) {}

  // Functor function that calculates the number of movies that can be scheduled for a given combination
  __device__ __host__
  int operator()(int combination) {
    bool timeSlots[24] = {false};  // Available time slots

    // Copy the category limits to a local array
    int localCategoryLimits[20];
    for (int i = 0; i <= categoryCount; i++) {
      localCategoryLimits[i] = categoryLimits[i];
    }

    int scheduledMovies = 0;  // Counter for scheduled movies
    for (int i = 0; i < movieCount; i++) {
      if (combination & (1 << i)) {  // Check if the i-th movie is included in the combination
        Movie& currentMovie = movies[i];

        // Check if time slots for the movie's duration are occupied
        for (int j = currentMovie.start; j < currentMovie.end; j++) {
          if (timeSlots[j]) return -1;  // If a time slot is occupied, return -1
          timeSlots[j] = true;  // Mark time slot as occupied
        }
        // If category limit is reached, return -1
        if (localCategoryLimits[currentMovie.category] == 0) return -1;
        localCategoryLimits[currentMovie.category]--;  // Decrease the limit for the movie's category
        scheduledMovies++;
      }
    }
    return scheduledMovies;
  }
};

// Main function
int main(int argc, char *argv[]) {
  // Get number of movies and categories from the user
  int num_movies, num_categories;
  cin >> num_movies >> num_categories;

  // Get the limit for each category
  vector<int> category_limit(num_categories + 1);
  for (int i = 1; i <= num_categories; i++) {
    cin >> category_limit[i];
  }

  // Get the movie details
  vector<Movie> movies(num_movies);
  for (int i = 0; i < num_movies; i++) {
    cin >> movies[i].start >> movies[i].end >> movies[i].category;
    if (movies[i].end < movies[i].start) {
      movies[i].end = 24;
    }

    if (movies[i].end == movies[i].start) {
      movies[i].end += 1;
    }
  }

  // Start the timer
  auto startTime = chrono::steady_clock::now();
  
  // Move data to GPU
  int *category_limit_gpu;
  Movie *movies_gpu;
  hipMalloc(&category_limit_gpu, category_limit.size() * sizeof(int));
  hipMalloc(&movies_gpu, movies.size() * sizeof(Movie));
  hipMemcpy(category_limit_gpu, category_limit.data(), category_limit.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(movies_gpu, movies.data(), movies.size() * sizeof(Movie), hipMemcpyHostToDevice);
  
  thrust::device_vector<int> movie_counts(pow(2, movies.size()));
  thrust::counting_iterator<int> combinations(0);

  ScheduleFunctor functor(
    movies.size(), 
    movies_gpu, 
    num_categories, 
    category_limit_gpu
  );

  // Apply Functor to each element in the vector
  thrust::transform(combinations, combinations + pow(2, movies.size()), movie_counts.begin(), functor);

  // Find the maximum number of movies
//   int max_movies = thrust::reduce(movie_counts.begin(), movie_counts.end(), thrust::maximum<int>())
  int max_movies = *thrust::max_element(movie_counts.begin(), movie_counts.end());

  // Print the result
  cout << "\nNúmero de filmes: " << max_movies << "\n";
  
  // Stop the timer and calculate the elapsed time
  auto endTime = chrono::steady_clock::now();
  double duration = chrono::duration_cast<chrono::microseconds>(endTime - startTime).count();

  // Print the elapsed time
  cout << fixed;
  cout << "Time elapsed during the brute-force algorithm (in microseconds): " << duration << endl;
  cout << scientific;

  hipFree(category_limit_gpu);
  hipFree(movies_gpu);

  return 0;
}
