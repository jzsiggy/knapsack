#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <thrust/device_vector.h>

int main() {
    auto startTime = std::chrono::steady_clock::now();

    int num_movies, num_categories;
    std::cin >> num_movies >> num_categories;

    std::vector<int> h_start_times(num_movies);
    std::vector<int> h_end_times(num_movies);
    std::vector<int> h_categories(num_movies);
    std::vector<int> h_L(num_categories);

    for (int i = 0; i < num_movies; i++) {
        std::cin >> h_start_times[i] >> h_end_times[i] >> h_categories[i];
        if (h_end_times[i] < h_start_times[i]) {
            h_end_times[i] = 24;
        }
        if (h_end_times[i] == h_start_times[i]) {
            h_end_times[i] += 1;
        }
        h_categories[i]--;  // convert categories to 0-based index
    }

    for (int i = 0; i < num_categories; i++) {
        std::cin >> h_L[i];
    }

    // Copy host vectors to device
    thrust::device_vector<int> start_times = h_start_times;
    thrust::device_vector<int> end_times = h_end_times;
    thrust::device_vector<int> categories = h_categories;
    thrust::device_vector<int> L = h_L;

    thrust::device_vector<int> dp((num_movies+1) * (num_categories+1), 0);

    for (int i = 1; i <= num_movies; i++) {
        for (int j = 1; j <= num_categories; j++) {
            int max_count = 0;
            for (int k = 0; k < i; k++) {
                if (categories[k] == j-1 && end_times[k] <= start_times[i-1] && dp[k * (num_categories+1) + j-1] + 1 <= L[j-1]) {
                    max_count = std::max(max_count, (int)dp[k * (num_categories+1) + j-1] + 1);
                } else {
                    max_count = std::max(max_count, (int)dp[k * (num_categories+1) + j]);
                }
            }
            dp[i * (num_categories+1) + j] = max_count;
        }
    }

    int max_count = 0;
    for (int j = 1; j <= num_categories; j++) {
        max_count = std::max(max_count, (int)dp[num_movies * (num_categories+1) + j]);
    }

    std::cout << "\nNúmero de filmes: " << max_count << "\n";

    auto endTime = std::chrono::steady_clock::now();
    double duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();

    std::cout << std::fixed;
    std::cout << "Time elapsed during the brute-force algorithm (in microseconds): " << duration << std::endl;
    std::cout << std::scientific;

    return 0;
}
